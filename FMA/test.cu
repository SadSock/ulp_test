#include "hip/hip_runtime.h"
#include <cstdint>
#include <stdio.h>
#include <stdint.h>
#include <mpfr.h>

// 定义向量长度
#define N 4

// CUDA 核函数，用于计算两个向量的点积
__global__ void dot_product(double *a, double *b, double *result) {
    result[0] = 0.0f;
    for(int i = 0; i < 4; i++){
        result[0] += a[i] * b[i];
    }    
}

int main() {
    double a[N] = {0.0f};
    double b[N] = {0.0f};
    double GPU_ret[N] = {0.0f};


    // 使用当前时间作为随机数生成器的种子
    srand(time(NULL));
    // 分配 GPU 内存
    double *dev_a, *dev_b, *dev_result;
    hipMalloc((void**)&dev_a, N * sizeof(double));
    hipMalloc((void**)&dev_b, N * sizeof(double));
    hipMalloc((void**)&dev_result, N * sizeof(double));
    size_t cnt = 0;
            while(cnt < 100000){
        // 生成8个随机数，并将其存储在数组中
    uint64_t num;
        for (int i = 0; i < 4; i++) {
            num = (((uint64_t)rand()) << 32) | rand();
            *(uint64_t *)&a[i] = num;
            num = (((uint64_t)rand()) << 32) | rand();
            *(uint64_t *)&b[i] = num;
        }
            //// 打印读取的值
            //for (int i = 0; i < 4; i++) {
              //printf("a[%d] = %la\n", i, a[i]);
            //}
            //for (int i = 0; i < 4; i++) {
              //printf("b[%d] = %la\n", i, b[i]);
            //}

        // 将数据复制到 GPU 内存中
        hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

        // 定义块大小和线程大小
        dim3 block_size(1, 1, 1);
        dim3 grid_size(1, 1, 1);

        // 调用 CUDA 核函数
        dot_product<<<grid_size, block_size>>>(dev_a, dev_b, dev_result);

        // 将结果从 GPU 内存复制回 CPU 内存
        hipMemcpy(GPU_ret, dev_result, N * sizeof(double),
                   hipMemcpyDeviceToHost);

        // CPU计算点积并打印结果
        double cpu_ret = 0.0f;
        for (int i = 0; i < 4; i++) {
            cpu_ret += a[i] * b[i];
        }

    if(isinf(cpu_ret) || isinf(GPU_ret[0]))
        continue;

  mpfr_t ma[N], mb[N], mpfr_ret;
  mpfr_t ma_gpu, ma_cpu;
  mpfr_t ma_dgpu, ma_dcpu;
    
    mpfr_init2(ma_gpu, 1024);
    mpfr_set_d(ma_gpu, GPU_ret[0], MPFR_RNDN);
    
    mpfr_init2(ma_cpu, 1024);
    mpfr_set_d(ma_cpu, cpu_ret, MPFR_RNDN);
  
        
  for (int i = 0; i < N; i++) {
    mpfr_init2(ma[i], 1024);
    mpfr_set_d(ma[i], a[i], MPFR_RNDN);
  }

  
  for (int i = 0; i < N; i++) {
    mpfr_init2(mb[i], 1024);
    mpfr_set_d(mb[i], b[i], MPFR_RNDN);
  }

  mpfr_init2(mpfr_ret, 1024);
  mpfr_set_d(mpfr_ret, 0.0, MPFR_RNDN);

  for (int i = 0; i < N; i++) {
    mpfr_fma(mpfr_ret, ma[i], mb[i], mpfr_ret, MPFR_RNDN);
  }


    mpfr_init2(ma_dcpu, 1024);
    mpfr_init2(ma_dgpu, 1024);
    mpfr_sub(ma_dgpu, ma_gpu, mpfr_ret, MPFR_RNDN);
    mpfr_sub(ma_dcpu, ma_cpu, mpfr_ret, MPFR_RNDN);
    
    mpfr_abs(ma_dgpu, ma_dgpu, MPFR_RNDN);
    mpfr_abs(ma_dcpu, ma_dcpu, MPFR_RNDN);


 int cmp = mpfr_cmp(ma_dgpu, ma_dcpu);
    if (cmp < 0) {
        printf("GPU win\n");
    } else if (cmp > 0) {
        printf("cpu win\n");
    } else {
        printf("double win\n");
    }

  mpfr_printf("mpfr %.100RNf\n", mpfr_ret);
  printf("X86  %f\n", cpu_ret);
  printf("cuda %f\n", GPU_ret[0]);
cnt = cnt + 1;
    }
    // 释放 GPU 内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
	}
