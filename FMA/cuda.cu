
#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>
#include <stdint.h>

// 定义向量长度
#define N 4

// CUDA 核函数，用于计算两个向量的点积
__global__ void dot_product(double *a, double *b, double *result) {
    result[0] = 0.0f;
    for(int i = 0; i < 4; i++){
        result[0] += a[i] * b[i];
        printf("%lx, %la\n", *(uint64_t*)&result[0], result[0]);
    }    
}

int main() {
    double a[N] = {0.0f};
    double b[N] = {0.0f};
    double result[N] = {0.0f};

    FILE *fp;

    fp = fopen("input.txt", "r");
    if (fp == NULL) {
        printf("Error opening file\n");
        return 1;
    }

    for (int i = 0; i < 4; i++) {
        if (fscanf(fp, "%la", &a[i]) != 1) {
          printf("Error reading file\n");
          return 1;
        }
        printf("%la\n", a[i]);
    }

    for (int i = 0; i < 4; i++) {
        if (fscanf(fp, "%la", &b[i]) != 1) {
          printf("Error reading file\n");
          return 1;
        }
        printf("%la\n", b[i]);
    }

    fclose(fp);
    // 分配 GPU 内存
    double *dev_a, *dev_b, *dev_result;
    hipMalloc((void **)&dev_a, N * sizeof(double));
    hipMalloc((void **)&dev_b, N * sizeof(double));
    hipMalloc((void **)&dev_result, N * sizeof(double));
    // 将数据复制到 GPU 内存中
    hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    // 定义块大小和线程大小
    dim3 block_size(1, 1, 1);
    dim3 grid_size(1, 1, 1);

    // 调用 CUDA 核函数
    dot_product<<<grid_size, block_size>>>(dev_a, dev_b, dev_result);

    // 将结果从 GPU 内存复制回 CPU 内存
    hipMemcpy(result, dev_result, N * sizeof(double), hipMemcpyDeviceToHost);

    // 计算点积并打印结果
    printf("cuda %0.100f %la\n", result[0], result[0]);

    // 释放 GPU 内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
}
