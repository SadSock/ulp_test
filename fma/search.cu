#include "hip/hip_runtime.h"
#include <cstdint>
#include <stdio.h>
#include <stdint.h>

// 定义向量长度
#define N 4

// CUDA 核函数，用于计算两个向量的点积
__global__ void dot_product(double *a, double *b, double *result) {
    result[0] = 0.0f;
    for(int i = 0; i < 4; i++){
        result[0] += a[i] * b[i];
    }    
}

int main() {
    double a[N] = {0.0f};
    double b[N] = {0.0f};
    double result[N] = {0.0f};

    FILE *fp;
    uint64_t num;

    fp = fopen("input.txt", "r");
    if (fp == NULL) {
        printf("Error opening file\n");
        return 1;
    }

    for (int i = 0; i < 4; i++) {
        if (fscanf(fp, "%la", &a[i]) != 1) {
            printf("Error reading file\n");
            return 1;
        }
    }

    for (int i = 0; i < 4; i++) {
        if (fscanf(fp, "%la", &b[i]) != 1) {
            printf("Error reading file\n");
            return 1;
        }
    }

    // 使用当前时间作为随机数生成器的种子
    srand(time(NULL));
    fclose(fp);
    // 分配 GPU 内存
    double *dev_a, *dev_b, *dev_result;
    hipMalloc((void**)&dev_a, N * sizeof(double));
    hipMalloc((void**)&dev_b, N * sizeof(double));
    hipMalloc((void**)&dev_result, N * sizeof(double));
    for (int j = 0; j < 100; j++){
        // 生成8个随机数，并将其存储在数组中
        for (int i = 0; i < 4; i++) {
            num = (((uint64_t)rand()) << 32) | rand();
            *(uint64_t *)&a[i] = num;
            num = (((uint64_t)rand()) << 32) | rand();
            *(uint64_t *)&b[i] = num;
        }
        // 将数据复制到 GPU 内存中
        hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

        // 定义块大小和线程大小
        dim3 block_size(1, 1, 1);
        dim3 grid_size(1, 1, 1);

        // 调用 CUDA 核函数
        dot_product<<<grid_size, block_size>>>(dev_a, dev_b, dev_result);

        // 将结果从 GPU 内存复制回 CPU 内存
        hipMemcpy(result, dev_result, N * sizeof(double),
                   hipMemcpyDeviceToHost);

        // 计算点积并打印结果
        double dot_product = 0.0f;
        for (int i = 0; i < 4; i++) {
            dot_product += a[i] * b[i];
        }

        if(dot_product != result[0]){
            printf("X86  %f\n", dot_product);
            printf("cuda %f\n", result[0]);

            // 打印读取的值
            for (int i = 0; i < 4; i++) {
              printf("a[%d] = %la\n", i, a[i]);
            }
            for (int i = 0; i < 4; i++) {
              printf("b[%d] = %la\n", i, b[i]);
            }
        }
    }
    // 释放 GPU 内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
	}
