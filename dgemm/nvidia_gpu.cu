#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


int main(void)
{
    const int n = 1000;
    const int k = 1000;
    const int m = 1000;

    double *A, *B, *C;
    double alpha = 1.0, beta = 0.0;

    A = (double*) malloc(n * k * sizeof(double));
    B = (double*) malloc(k * m * sizeof(double));
    C = (double*) malloc(n * m * sizeof(double));

    // 从文件中读取矩阵 A 和 B 的数据
    FILE* fp = fopen("data.bin", "rb");
    fread(A, sizeof(double), n * k, fp);
    fread(B, sizeof(double), k * m, fp);
    fclose(fp);

    // 在 C 中填充零值
    for (int i = 0; i < n * m; ++i) {
        C[i] = 0.0;
    }

    double* d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * k * sizeof(double));
    hipMalloc(&d_B, k * m * sizeof(double));
    hipMalloc(&d_C, n * m * sizeof(double));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMemcpy(d_A, A, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * m * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, n * m * sizeof(double), hipMemcpyHostToDevice);

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_B, m, d_A, k, &beta, d_C, m);

    hipMemcpy(C, d_C, n * m * sizeof(double), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);


    fp = fopen("nvidia_gpu.bin", "wb");
    fwrite(C, sizeof(double), n * m, fp);
    fclose(fp);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
