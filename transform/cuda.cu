
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <math.h>

__global__ void double2uint64(double ret_f64) {
  uint64_t ret_u64;
  // 从外部读入
  ret_u64 = ret_f64;
  printf("%lx\n", ret_u64);

  // 先赋值给double类型，再隐式转换为uint64_t
  ret_f64 = ldexp(1.0, 64);
  ret_u64 = ret_f64;
  printf("%lx\n", ret_u64);

  // 直接隐式转换并赋值给uint64_t
  ret_u64 = ldexp(1.0, 64);
  printf("%lx\n", ret_u64);
}

int main(void)
{
  double ret_f64 = ldexp(1.0, 64);
  // 启动CUDA内核函数
  double2uint64<<<1, 1>>>(ret_f64);
  hipDeviceSynchronize();
  return 0;
}
